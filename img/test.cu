#include <cstdlib>
#include <memory>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdio>
#include <time.h>
#include <fstream>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <typeinfo>
#include <unistd.h>


#define USE_MNIST_LOADER
#define MNIST_DOUBLE
#ifndef __MNIST_H__
#define __MNIST_H__

/*
 * MNIST loader by Nuri Park - https://github.com/projectgalateia/mnist
 */

#ifdef USE_MNIST_LOADER /* Fundamental macro to make the code active */

#ifdef __cplusplus
extern "C" {
#endif

/*
 * Make mnist_load function static.
 * Define when the header is included multiple time.
 */
#ifdef MNIST_STATIC
#define _STATIC static
#else
#define _STATIC 
#endif

/*
 * Make mnist loader to load image data as double type.
 * It divides unsigned char values by 255.0, so the results ranges from 0.0 to 1.0
 */
#ifdef MNIST_DOUBLE
#define MNIST_DATA_TYPE double
#else
#define MNIST_DATA_TYPE unsigned char
#endif

typedef struct mnist_data {
	MNIST_DATA_TYPE data[28][28]; /* 28x28 data for the image */
	unsigned int label; /* label : 0 to 9 */
} mnist_data;

/*
 * If it's header inclusion, make only function prototype visible.
 */
#ifdef MNIST_HDR_ONLY

_STATIC int mnist_load(
	const char *image_filename,
	const char *label_filename,
	mnist_data **data,
	unsigned int *count);

#else



/*
 * Load a unsigned int from raw data.
 * MSB first.
 */
static unsigned int mnist_bin_to_int(char *v)
{
	int i;
	unsigned int ret = 0;

	for (i = 0; i < 4; ++i) {
		ret <<= 8;
		ret |= (unsigned char)v[i];
	}

	return ret;
}

/*
 * MNIST dataset loader.
 *
 * Returns 0 if successed.
 * Check comments for the return codes.
 */
_STATIC int mnist_load(
	const char *image_filename,
	const char *label_filename,
	mnist_data **data,
	unsigned int *count)
{
	int return_code = 0;
	int i;
	char tmp[4];

	unsigned int image_cnt, label_cnt;
	unsigned int image_dim[2];

	FILE *ifp = fopen(image_filename, "rb");
	FILE *lfp = fopen(label_filename, "rb");

	if (!ifp || !lfp) {
		return_code = -1; /* No such files */
		goto cleanup;
	}

	fread(tmp, 1, 4, ifp);
	if (mnist_bin_to_int(tmp) != 2051) {
		return_code = -2; /* Not a valid image file */
		goto cleanup;
	}

	fread(tmp, 1, 4, lfp);
	if (mnist_bin_to_int(tmp) != 2049) {
		return_code = -3; /* Not a valid label file */
		goto cleanup;
	}

	fread(tmp, 1, 4, ifp);
	image_cnt = mnist_bin_to_int(tmp);

	fread(tmp, 1, 4, lfp);
	label_cnt = mnist_bin_to_int(tmp);

	if (image_cnt != label_cnt) {
		return_code = -4; /* Element counts of 2 files mismatch */
		goto cleanup;
	}

	for (i = 0; i < 2; ++i) {
		fread(tmp, 1, 4, ifp);
		image_dim[i] = mnist_bin_to_int(tmp);
	}

	if (image_dim[0] != 28 || image_dim[1] != 28) {
		return_code = -2; /* Not a valid image file */
		goto cleanup;
	}

	*count = image_cnt;
	*data = (mnist_data *)malloc(sizeof(mnist_data) * image_cnt);

	for (i = 0; i < image_cnt; ++i) {
		int j;
		unsigned char read_data[28 * 28];
		mnist_data *d = &(*data)[i];

		fread(read_data, 1, 28*28, ifp);

#ifdef MNIST_DOUBLE
		for (j = 0; j < 28*28; ++j) {
			d->data[j/28][j%28] = read_data[j] / 255.0;
		}
#else
		memcpy(d->data, read_data, 28*28);
#endif

		fread(tmp, 1, 1, lfp);
		d->label = tmp[0];
	}

cleanup:
	if (ifp) fclose(ifp);
	if (lfp) fclose(lfp);

	return return_code;
}

#endif /* MNIST_HDR_ONLY */

#ifdef __cplusplus
}
#endif

#endif /* USE_MNIST_LOADER */
#endif /* __MNIST_H__ */




const static float dt = 1.0E-01f;
const static float threshold = 1.0E-02f;
static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;



// define Layer class
class Layer {
	public:
	int M, N, O;

	float *output;
	float *preact;

	float *bias;
	float *weight;

	float *d_output;
	float *d_preact;
	float *d_weight;

	Layer(int M, int N, int O);

	~Layer();

	void setOutput(float *data);
	void clear();
	void bp_clear();
	//void save_param();
};



// Utility CUDA kernel functions
__device__ float step_function(float v);
__global__ void apply_step_function(float *input, float *output, const int N);
__global__ void makeError(float *err, float *output, unsigned int Y, const int N);
__global__ void apply_grad(float *output, float *grad, const int N);

// Forward propagation kernels
__global__ void fp_preact_c1(float input[28][28], float preact[6][24][24], float weight[6][5][5]);
__global__ void fp_bias_c1(float preact[6][24][24], float bias[6]);
__global__ void fp_preact_s1(float input[6][24][24], float preact[6][6][6], float weight[1][4][4]);
__global__ void fp_bias_s1(float preact[6][6][6], float bias[1]);
__global__ void fp_preact_f(float input[6][6][6], float preact[10], float weight[10][6][6][6]);
__global__ void fp_bias_f(float preact[10], float bias[10]);

// Back propagation kernels
__global__ void bp_weight_f(float d_weight[10][6][6][6], float d_preact[10], float p_output[6][6][6]);
__global__ void bp_bias_f(float bias[10], float d_preact[10]);
__global__ void bp_output_s1(float d_output[6][6][6], float n_weight[10][6][6][6], float nd_preact[10]);
__global__ void bp_preact_s1(float d_preact[6][6][6], float d_output[6][6][6], float preact[6][6][6]);
__global__ void bp_weight_s1(float d_weight[1][4][4], float d_preact[6][6][6], float p_output[6][24][24]);
__global__ void bp_bias_s1(float bias[1], float d_preact[6][6][6]);
__global__ void bp_output_c1(float d_output[6][24][24], float n_weight[1][4][4], float nd_preact[6][6][6]);
__global__ void bp_preact_c1(float d_preact[6][24][24], float d_output[6][24][24], float preact[6][24][24]);
__global__ void bp_weight_c1(float d_weight[6][5][5], float d_preact[6][24][24], float p_output[28][28]);
__global__ void bp_bias_c1(float bias[6], float d_preact[6][24][24]);


// Constructor
Layer::Layer(int M, int N, int O)
{
	this->M = M;
	this->N = N;
	this->O = O;

	float h_bias[N];
	float h_weight[N][M];

	output = NULL;
	preact = NULL;
	bias   = NULL;
	weight = NULL;

	for (int i = 0; i < N; ++i) {
		h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);

		for (int j = 0; j < M; ++j) {
			h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
		}
	}

	hipMalloc(&output, sizeof(float) * O);
	hipMalloc(&preact, sizeof(float) * O);

	hipMalloc(&bias, sizeof(float) * N);

	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * O);
	hipMalloc(&d_preact, sizeof(float) * O);
	hipMalloc(&d_weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);

	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// Destructor
Layer::~Layer()
{
	hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);
}

// Send data one row from dataset to the GPU
void Layer::setOutput(float *data)
{
	hipMemcpy(output, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer::clear()
{
	hipMemset(output, 0x00, sizeof(float) * O);
	hipMemset(preact, 0x00, sizeof(float) * O);
}

void Layer::bp_clear()
{
	hipMemset(d_output, 0x00, sizeof(float) * O);
	hipMemset(d_preact, 0x00, sizeof(float) * O);
	hipMemset(d_weight, 0x00, sizeof(float) * M * N);
}


__device__ float step_function(float v)
{
	return 1 / (1 + exp(-v));
}

__global__ void apply_step_function(float *input, float *output, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] = step_function(input[idx]);
	}
}

__global__ void custom_snrm2(float* x, int n, float* result)
{
    __shared__ float cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float sum = 0;
    while (tid < n) {
        sum += x[tid] * x[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = sum;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        atomicAdd(result, sqrt(cache[0]));
    }
}

__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
	}
}

__global__ void apply_grad(float *output, float *grad, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] += dt * grad[idx];
	}
}

__global__ void fp_preact_c1(float input[28][28], float preact[6][24][24], float weight[6][5][5])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 5*5*6*24*24;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 5);
		const int i2 = ((idx /= 5	) % 5);
		const int i3 = ((idx /= 5	) % 6);
		const int i4 = ((idx /= 6	) % 24);
		const int i5 = ((idx /= 24	) % 24);

		atomicAdd(&preact[i3][i4][i5], weight[i3][i1][i2] * input[i4 + i1][i5 + i2]);
	}
}

__global__ void fp_bias_c1(float preact[6][24][24], float bias[6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 24);
		const int i3 = ((idx /= 24	) % 24);

		preact[i1][i2][i3] += bias[i1];
	}
}


__global__ void fp_preact_s1(float input[6][24][24], float preact[6][6][6], float weight[1][4][4])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 4*4*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 4);
		const int i2 = ((idx /= 4	) % 4);
		const int i3 = ((idx /= 4	) % 6);
		const int i4 = ((idx /= 6	) % 6);
		const int i5 = ((idx /= 6	) % 6);

		atomicAdd(&preact[i3][i4][i5], weight[0][i1][i2] * input[i3][i4 * 4 + i1][i5 * 4 + i2]);
	}
}

__global__ void fp_bias_s1(float preact[6][6][6], float bias[1])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 6);
		const int i3 = ((idx /= 6	) % 6);

		preact[i1][i2][i3] += bias[0];
	}
}

__global__ void fp_preact_f(float input[6][6][6], float preact[10], float weight[10][6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10	) % 6);
		const int i3 = ((idx /= 6	) % 6);
		const int i4 = ((idx /= 6	) % 6);

		atomicAdd(&preact[i1], weight[i1][i2][i3][i4] * input[i2][i3][i4]);
	}
}

__global__ void fp_bias_f(float preact[10], float bias[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		preact[idx] += bias[idx];
	}
}

__global__ void bp_weight_f(float d_weight[10][6][6][6], float d_preact[10], float p_output[6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10	) % 6);
		const int i3 = ((idx /= 6	) % 6);
		const int i4 = ((idx /= 6	) % 6);

		d_weight[i1][i2][i3][i4] = d_preact[i1] * p_output[i2][i3][i4];
	}
}

__global__ void bp_bias_f(float bias[10], float d_preact[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		bias[idx] += dt * d_preact[idx];
	}
}

__global__ void bp_output_s1(float d_output[6][6][6], float n_weight[10][6][6][6], float nd_preact[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10	) % 6);
		const int i3 = ((idx /= 6	) % 6);
		const int i4 = ((idx /= 6	) % 6);

		atomicAdd(&d_output[i2][i3][i4], n_weight[i1][i2][i3][i4] * nd_preact[i1]);
	}
}

__global__ void bp_preact_s1(float d_preact[6][6][6], float d_output[6][6][6], float preact[6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 6);
		const int i3 = ((idx /= 6	) % 6);

		const float o = step_function(preact[i1][i2][i3]);

		d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
	}
}

__global__ void bp_weight_s1(float d_weight[1][4][4], float d_preact[6][6][6], float p_output[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 1*4*4*6*6*6;
	const float d = pow(6.0f, 3.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 1);
		const int i2 = ((idx /= 1	) % 4);
		const int i3 = ((idx /= 4	) % 4);
		const int i4 = ((idx /= 4	) % 6);
		const int i5 = ((idx /= 6	) % 6);
		const int i6 = ((idx /= 6	) % 6);

		atomicAdd(&d_weight[i1][i2][i3], d_preact[i4][i5][i6] * p_output[i4][i5 * 4 + i2][i6 * 4 + i3]);
	}
}

__global__ void bp_bias_s1(float bias[1], float d_preact[6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*6*6;
	const float d = pow(6.0f, 3.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 6);
		const int i3 = ((idx /= 6	) % 6);

		atomicAdd(&bias[0], dt * d_preact[i1][i2][i3] / d);
	}
}

__global__ void bp_output_c1(float d_output[6][24][24], float n_weight[1][4][4], float nd_preact[6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 1*4*4*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 1);
		const int i2 = ((idx /= 1	) % 4);
		const int i3 = ((idx /= 4	) % 4);
		const int i4 = ((idx /= 4	) % 6);
		const int i5 = ((idx /= 6	) % 6);
		const int i6 = ((idx /= 6	) % 6);

		atomicAdd(&d_output[i4][i5 * 4 + i2][i6 * 4 + i3], n_weight[i1][i2][i3] * nd_preact[i4][i5][i6]);
	}
}

__global__ void bp_preact_c1(float d_preact[6][24][24], float d_output[6][24][24], float preact[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 24);
		const int i3 = ((idx /= 24	) % 24);

		const float o = step_function(preact[i1][i2][i3]);

		d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
	}
}

__global__ void bp_weight_c1(float d_weight[6][5][5], float d_preact[6][24][24], float p_output[28][28])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*5*5*24*24;
	const float d = pow(24.0f, 2.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 5);
		const int i3 = ((idx /= 5	) % 5);
		const int i4 = ((idx /= 5	) % 24);
		const int i5 = ((idx /= 24	) % 24);

		atomicAdd(&d_weight[i1][i2][i3], d_preact[i1][i4][i5] * p_output[i4 + i2][i5 + i3] / d);
	}
}

__global__ void bp_bias_c1(float bias[6], float d_preact[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24;
	const float d = pow(24.0f, 2.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 24);
		const int i3 = ((idx /= 24	) % 24);

		atomicAdd(&bias[i1], dt * d_preact[i1][i2][i3] / d);
	}
}

static Layer l_input = Layer(0, 0, 28*28);
static Layer l_c1 = Layer(5*5, 6, 24*24*6);
static Layer l_s1 = Layer(4*4, 1, 6*6*6);
static Layer l_f = Layer(6*6*6, 10, 10);

static void learn();
static unsigned int classify(double data[28][28]);
static double test();
static double forward_pass(double data[28][28]);
static double back_pass();
void save_params(Layer* layer,const char* filename);
static void load_params(Layer* layer, char* filename);
static void learn_with_save(std::string dir);
static void test_with_load(std::string dir);

int main(int argc, char* argv[])
{

	std::string dir = argv[1];
	std::string train_images_path = dir + "/../../data/FashionMNIST/raw/train-images-idx3-ubyte";
	std::string train_labels_path = dir + "/../../data/FashionMNIST/raw/train-labels-idx1-ubyte";
	std::string test_images_path = dir + "/../../data/FashionMNIST/raw/t10k-images-idx3-ubyte";
	std::string test_labels_path = dir + "/../../data/FashionMNIST/raw/t10k-labels-idx1-ubyte";
	int ret;
	if (ret = mnist_load(test_images_path.c_str(), test_labels_path.c_str(), &test_set, &test_cnt)) { 
			fflush(stderr);
			fprintf(stderr, "An error occured: %d\n", ret);
			}
	if (ret = mnist_load(train_images_path.c_str(), train_labels_path.c_str(), &train_set, &train_cnt)) {
			fflush(stderr);
			fprintf(stderr, "An error occured: %d\n", ret);
			}
	test_with_load(dir); // 貌似没读上参数，但是我在本地先跑train后跑test是可以的
	// learn();
	// auto start = std::chrono::high_resolution_clock::now();
	// double acc = test();
	// auto end = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double> diff = end - start;
	// fprintf(stdout, "%.4f:%.4f\n", diff.count(), acc);
	return 0;
}

static void test_with_load(std::string dir){
	std::string c1_param = dir + "/c1.0033";
	std::string s1_param = dir + "/s1.0033";
	std::string f_param = dir + "/f.0033";
	load_params(&l_c1, const_cast<char*>(c1_param.c_str()));
	load_params(&l_s1, const_cast<char*>(s1_param.c_str()));
	load_params(&l_f, const_cast<char*>(f_param.c_str()));
	// fprintf(stdout, "load params from %s, %s, %s\n", c1_param.c_str(), s1_param.c_str(), f_param.c_str());
	auto start = std::chrono::high_resolution_clock::now();
	double acc = test();
	auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
	fprintf(stdout, "%.4f:%.4f\n", diff.count(), acc);
}

// Forward propagation of a single row in dataset
static double forward_pass(double data[28][28])
{
	float input[28][28];

	for (int i = 0; i < 28; ++i) {
		for (int j = 0; j < 28; ++j) {
			input[i][j] = data[i][j];
		}
	}

	l_input.clear();
	l_c1.clear();
	l_s1.clear();
	l_f.clear();

	clock_t start, end;
	start = clock();

	l_input.setOutput((float *)input);
	
	fp_preact_c1<<<64, 64>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight);
	fp_bias_c1<<<64, 64>>>((float (*)[24][24])l_c1.preact, l_c1.bias);
	apply_step_function<<<64, 64>>>(l_c1.preact, l_c1.output, l_c1.O);

	fp_preact_s1<<<64, 64>>>((float (*)[24][24])l_c1.output, (float (*)[6][6])l_s1.preact, (float (*)[4][4])l_s1.weight);
	fp_bias_s1<<<64, 64>>>((float (*)[6][6])l_s1.preact, l_s1.bias);
	apply_step_function<<<64, 64>>>(l_s1.preact, l_s1.output, l_s1.O);

	fp_preact_f<<<64, 64>>>((float (*)[6][6])l_s1.output, l_f.preact, (float (*)[6][6][6])l_f.weight);
	fp_bias_f<<<64, 64>>>(l_f.preact, l_f.bias);
	apply_step_function<<<64, 64>>>(l_f.preact, l_f.output, l_f.O);
	
	end = clock();
	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

// Back propagation to update weights
static double back_pass()
{
	clock_t start, end;

	start = clock();

	bp_weight_f<<<64, 64>>>((float (*)[6][6][6])l_f.d_weight, l_f.d_preact, (float (*)[6][6])l_s1.output);
	bp_bias_f<<<64, 64>>>(l_f.bias, l_f.d_preact);

	bp_output_s1<<<64, 64>>>((float (*)[6][6])l_s1.d_output, (float (*)[6][6][6])l_f.weight, l_f.d_preact);
	bp_preact_s1<<<64, 64>>>((float (*)[6][6])l_s1.d_preact, (float (*)[6][6])l_s1.d_output, (float (*)[6][6])l_s1.preact);
	bp_weight_s1<<<64, 64>>>((float (*)[4][4])l_s1.d_weight, (float (*)[6][6])l_s1.d_preact, (float (*)[24][24])l_c1.output);
	bp_bias_s1<<<64, 64>>>(l_s1.bias, (float (*)[6][6])l_s1.d_preact);

	bp_output_c1<<<64, 64>>>((float (*)[24][24])l_c1.d_output, (float (*)[4][4])l_s1.weight, (float (*)[6][6])l_s1.d_preact);
	bp_preact_c1<<<64, 64>>>((float (*)[24][24])l_c1.d_preact, (float (*)[24][24])l_c1.d_output, (float (*)[24][24])l_c1.preact);
	bp_weight_c1<<<64, 64>>>((float (*)[5][5])l_c1.d_weight, (float (*)[24][24])l_c1.d_preact, (float (*)[28])l_input.output);
	bp_bias_c1<<<64, 64>>>(l_c1.bias, (float (*)[24][24])l_c1.d_preact);


	apply_grad<<<64, 64>>>(l_f.weight, l_f.d_weight, l_f.M * l_f.N);
	apply_grad<<<64, 64>>>(l_s1.weight, l_s1.d_weight, l_s1.M * l_s1.N);
	apply_grad<<<64, 64>>>(l_c1.weight, l_c1.d_weight, l_c1.M * l_c1.N);

	end = clock();
	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

// Unfold the input layer
static void unfold_input(double input[28][28], double unfolded[24*24][5*5])
{
	int a = 0;
	(void)unfold_input;

	for (int i = 0; i < 2; ++i)
		for (int j = 0; j < 2; ++j) {
			int b = 0;
			for (int x = i; x < i + 2; ++x)
				for (int y = j; y < j+2; ++y)
					unfolded[a][b++] = input[x][y];
			a++;
		}
}

static void learn()
{

	float err;
	int iter = 1;
	
	double time_taken = 0.0;

	while (iter < 0 || iter-- > 0) {
		err = 0.0f;

		for (int i = 0; i < train_cnt; ++i) {
            float *tmp_err;
            float tmp_err_host;
            hipMalloc(&tmp_err, sizeof(float));
            hipMemset(tmp_err, 0, sizeof(float));

			time_taken += forward_pass(train_set[i].data);
			
			
			l_f.bp_clear();
			l_s1.bp_clear();
			l_c1.bp_clear();

			// Euclid distance of train_set[i]
			makeError<<<10, 1>>>(l_f.d_preact, l_f.output, train_set[i].label, 10);
            custom_snrm2<<<1, 10>>>(l_f.d_preact, 10, tmp_err); 
            hipMemcpy(&tmp_err_host, tmp_err, sizeof(float), hipMemcpyDeviceToHost);
			err += tmp_err_host;

			time_taken += back_pass();

		}
		


	}

}

// Returns label of given data (0-9)
static unsigned int classify(double data[28][28])
{
	float res[10];
	
	forward_pass(data);

	unsigned int max = 0;

	hipMemcpy(res, l_f.output, sizeof(float) * 10, hipMemcpyDeviceToHost);

	for (int i = 1; i < 10; ++i) {
		if (res[max] < res[i]) {
			max = i;
		}
	}

	return max;
}

// Perform forward propagation of test data
static double test()
{
	int error = 0;

	for (int i = 0; i < test_cnt; ++i) {
		if (classify(test_set[i].data) != test_set[i].label) {
			++error;
		}
	}

	return 1.0-(double(error) / double(test_cnt));
}

void save_params(Layer* layer, const char* filename){
	float bias_save[layer->N];
	float weight_save[layer->N][layer->M];
	hipMemcpy(bias_save, layer->bias, layer->N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy2D(weight_save, layer->M * sizeof(float), layer->weight, layer->M * sizeof(float), layer->M * sizeof(float), layer->N, hipMemcpyDeviceToHost);
	// fprintf(stdout, "file name: %s\n", filename);
	std::ofstream file(filename, std::ios::app);
		if (!file.is_open()) {
			fflush(stderr);
			fprintf(stderr, "无法打开文件\n");
			return;
		}

	for (auto bia : bias_save){
		file << bia << " ";
	}
	
	for (int i=0; i<layer->N; ++i) {
		for (int j=0; j<layer->M; ++j) {
			file << weight_save[i][j] << " ";
		}
	}
		
	file.close();
	return;
}

static void load_params(Layer* layer, char* filename){
	float bias_save[layer->N];
	float weight_save[layer->N][layer->M];

	fprintf(stdout, "file name: %s\n", filename);
	// open file
	std::ifstream file(filename);
		if (!file.is_open()) {
			fflush(stderr);
			fprintf(stderr, "无法打开文件\n");
			return;
		}

	// read bias
	for (int i=0; i < layer->N; ++i) {
		file >> bias_save[i];
	}

	// read weight
	for (int i=0; i < layer->N; ++i){
		for (int j=0; j<layer->M; ++j){
			file >> weight_save[i][j];
		}
	}

	hipMemcpy(layer->bias, bias_save, layer->N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy2D(layer->weight, layer->M * sizeof(float), weight_save, layer->M * sizeof(float), layer->M * sizeof(float), layer->N, hipMemcpyHostToDevice);

	
}